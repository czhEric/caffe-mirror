#include <algorithm>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/interp_layer.hpp"

namespace caffe {


template <typename Dtype>
void InterpLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    NOT_IMPLEMENTED;
}

template <typename Dtype>
void InterpLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(InterpLayer);


}  // namespace caffe
